#include "hip/hip_runtime.h"
#include "hairRemovalEngine.cuh"

__global__ void extractLChannelKernel(uchar* src, float* dst, uchar* dst2, int nx, int ny, int nz) {
    int x = threadIdx.x + DETECT_TILE_X * blockIdx.x;
    int y = threadIdx.y + DETECT_TILE_Y * blockIdx.y;

    for (int i = 0; i < DETECT_TILE_X; i += DETECT_TILE_Y / DETECT_UNROLL_Y) {
        // take pixel from DRAM
        uchar R = src[((y + i) * nx * nz) + (x * nz) + 0];
        uchar G = src[((y + i) * nx * nz) + (x * nz) + 1];
        uchar B = src[((y + i) * nx * nz) + (x * nz) + 2];

        // RGB to XYZ
        float r = fdividef((float)R, 255.0f);
        float g = fdividef((float)G, 255.0f);
        float b = fdividef((float)B, 255.0f);
        r = ((r > 0.04045f) ? __powf(fdividef(r + 0.055f, 1.055f), 2.4f) : fdividef(r, 12.92f)) * 100.0f;
        g = ((g > 0.04045f) ? __powf(fdividef(g + 0.055f, 1.055f), 2.4f) : fdividef(g, 12.92f)) * 100.0f;
        b = ((b > 0.04045f) ? __powf(fdividef(b + 0.055f, 1.055f), 2.4f) : fdividef(b, 12.92f)) * 100.0f;

        // XYZ to LAB
        float Y = fdividef(0.2126f * r + 0.7152f * g + 0.0722f * b, 100.0f);
        Y = (Y > 0.008856f) ? cbrtf(Y) : fmaf(7.787f, Y, 0.1379f);
        float L = fmaf(116.0f, Y, -16.0f) * 2.55f;

        // set pixel to DRAM
        dst[(y + i) * nx + x] = L;
        dst2[(y + i) * nx + x] = R;
        dst2[(y + i) * nx + x + nx * ny] = G;
        dst2[(y + i) * nx + x + 2 * nx * ny] = B;
    }
}

__global__ void padDataClampToBorderKernel(float* d_Dst, float* d_Src, int fftH, int fftW, int dataH, int dataW, int kernelH, int kernelW, int kernelY, int kernelX) {
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int borderH = dataH + kernelY;
    const int borderW = dataW + kernelX;

    if (y < fftH && x < fftW) {
        int dy, dx;
        if (y < dataH) {
            dy = y;
        }
        if (x < dataW) {
            dx = x;
        }
        if (y >= dataH && y < borderH) {
            dy = dataH - 1;
        }
        if (x >= dataW && x < borderW) {
            dx = dataW - 1;
        }
        if (y >= borderH) {
            dy = 0;
        }
        if (x >= borderW) {
            dx = 0;
        }
        d_Dst[y * fftW + x] = LOAD_FLOAT(dy * dataW + dx);
    }
}

__global__ void padKernelKernel(float* d_Dst, float* d_Src, int fftH, int fftW, int kernelH, int kernelW, int kernelY, int kernelX) {
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (y < kernelH && x < kernelW)
    {
        int ky = y - kernelY;
        if (ky < 0)
        {
            ky += fftH;
        }
        int kx = x - kernelX;
        if (kx < 0)
        {
            kx += fftW;
        }
        d_Dst[ky * fftW + kx] = LOAD_FLOAT(y * kernelW + x);
    }
}

__global__ void modulateAndNormalizeKernel(fComplex* d_Dst, fComplex* d_DataSrc, fComplex* d_KernelSrc, int dataSize, float c) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= dataSize) return;
    fComplex a = d_KernelSrc[i];
    fComplex b = d_DataSrc[i];
    fComplex d;
    mulAndScaleModified(a, b, c, d);
    d_Dst[i] = d;
}

__global__ void cubeReductionKernel(float* d_Src, uchar* d_Dst, int fftH, int fftW, int dataH, int dataW, int depth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dataW && y < dataH) {
        float max_value = 0.0f;
        float current_value = 0.0f;
        int offset = 0;
        for (int i = 0; i < depth; i++) {
            offset = i * fftH * fftW;
            current_value = d_Src[y * fftW + x + offset];
            if (current_value > max_value) {
                max_value = current_value;
            }
        }
        d_Dst[y * dataW + x] = (uchar)min(max_value, 255.0f);
    }
}

__global__ void binarizeKernel(uchar* d_Src, int width, int height, int threshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    d_Src[y * width + x] = d_Src[y * width + x] >= threshold ? MAX_DYNAMIC_VALUE : 0;
}

__global__ void NaiveDilationKernel(uchar* d_Src, bool* d_Dst, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    if (d_Src[y * width + x] == MAX_DYNAMIC_VALUE) {
        d_Dst[max(y - 1, 0) * width + x] = 1; // up
        d_Dst[min(height - 1, y + 1) * width + x] = 1; // down
        d_Dst[y * width + max(x - 1, 0)] = 1;  // left
        d_Dst[y * width + min(width - 1, x + 1)] = 1; // right
    }

    __syncthreads();

    if (d_Dst[y * width + x] == 1) {
        d_Dst[max(y - 1, 0) * width + x] = 1; // up
        d_Dst[min(height - 1, y + 1) * width + x] = 1; // down
        d_Dst[y * width + max(x - 1, 0)] = 1;  // left
        d_Dst[y * width + min(width - 1, x + 1)] = 1; // right
    }
}

__global__ void makeMaskSrcImageKernel(uchar* src, bool* mask, float* maskedSrc, float max, float min, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    float value = ((float)src[i] - min) / (max - min);
    maskedSrc[i] = mask[i] ? 1.0f : value;
}

__global__ void make8UDstKernel(float* src, uchar* dst, float maxR, float minR, float maxG, float minG, float maxB, float minB, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    dst[(y * width * 3) + (x * 3) + 0] = (uchar)(src[y * width + x] * (maxR - minR) + minR);
    dst[(y * width * 3) + (x * 3) + 1] = (uchar)(src[y * width + x + width * height] * (maxG - minG) + minG);
    dst[(y * width * 3) + (x * 3) + 2] = (uchar)(src[y * width + x + width * height * 2] * (maxB - minB) + minB);
}

__global__ void NotKernel(bool* mask, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;
    mask[i] = !mask[i];
}

__global__ void pdeHeatDiffusionSMEMKernel(bool* mask, float* src, float* dst, int width, int height) {
    __shared__ float smem[(INPAINT_SMEM_TILE_X + 2) * STEP][(INPAINT_SMEM_TILE_Y + 2) * STEP];
    const int x = blockIdx.x * STEP * INPAINT_SMEM_TILE_X + threadIdx.x - INPAINT_SMEM_TILE_X;
    const int y = blockIdx.y * STEP * INPAINT_SMEM_TILE_Y + threadIdx.y - INPAINT_SMEM_TILE_Y;;

    // locate at each block (a thread block map into src pointer)
    dst += y * width + x;
    src += y * width + x;

    // put into active space
    for (int yy = 1; yy < 1 + STEP; yy++) {
        for (int xx = 1; xx < 1 + STEP; xx++) {
            smem[yy * INPAINT_SMEM_TILE_Y + threadIdx.y][xx * INPAINT_SMEM_TILE_X + threadIdx.x]
                = dst[yy * INPAINT_SMEM_TILE_Y * width + xx * INPAINT_SMEM_TILE_X];
        }
    }

    // corner space
    smem[threadIdx.y][threadIdx.x] = dst[1 * INPAINT_SMEM_TILE_Y * width + 1 * INPAINT_SMEM_TILE_X];
    smem[threadIdx.y][(1 + STEP) * INPAINT_SMEM_TILE_X + threadIdx.x] = dst[1 * INPAINT_SMEM_TILE_Y * width + STEP * INPAINT_SMEM_TILE_X];
    smem[(1 + STEP) * INPAINT_SMEM_TILE_Y + threadIdx.y][threadIdx.x] = dst[STEP * INPAINT_SMEM_TILE_Y * width + 1 * INPAINT_SMEM_TILE_X];
    smem[(1 + STEP) * INPAINT_SMEM_TILE_Y + threadIdx.y][(1 + STEP) * INPAINT_SMEM_TILE_X + threadIdx.x] = dst[STEP * INPAINT_SMEM_TILE_Y * width + STEP * INPAINT_SMEM_TILE_X];

    // put into left space
    for (int yy = 1; yy < STEP + 1; yy++) {
        //if (y < height - BlockDim_y * (1 + Step) && y >= 0)
        //    printf("%d - %d\n", y + yy * BlockDim_y, x);
        smem[yy * INPAINT_SMEM_TILE_Y + threadIdx.y][threadIdx.x] 
            = (x >= 0) ? dst[yy * INPAINT_SMEM_TILE_Y * width] : 0;
    }

    // put into right space
    for (int yy = 1; yy < STEP + 1; yy++) {
        smem[yy * INPAINT_SMEM_TILE_Y + threadIdx.y][(1 + STEP) * INPAINT_SMEM_TILE_X + threadIdx.x]
            = (x < width - (1 + STEP) * INPAINT_SMEM_TILE_X) ? dst[yy * INPAINT_SMEM_TILE_Y * width + (1 + STEP) * INPAINT_SMEM_TILE_X] : 0;
    }

    // put into top space
    for (int xx = 1; xx < STEP + 1; xx++) {
        smem[threadIdx.y][xx * INPAINT_SMEM_TILE_X + threadIdx.x]
            = (y >= 0) ? dst[xx * INPAINT_SMEM_TILE_X] : 0;
    }

    // put into bottom space
    for (int xx = 1; xx < STEP + 1; xx++) {
        smem[(1 + STEP) * INPAINT_SMEM_TILE_Y + threadIdx.y][xx * INPAINT_SMEM_TILE_X + threadIdx.x]
            = (y < height - (1 + STEP) * INPAINT_SMEM_TILE_Y) ? dst[(1 + STEP) * INPAINT_SMEM_TILE_Y * width + xx * INPAINT_SMEM_TILE_X] : 0;
    }
    __syncthreads();


    for (int yy = 1; yy < 1 + STEP; yy++) {
        for (int xx = 1; xx < 1 + STEP; xx++) {
            int index = yy * INPAINT_SMEM_TILE_Y * width + xx * INPAINT_SMEM_TILE_X;
            float center = smem[yy * INPAINT_SMEM_TILE_Y + threadIdx.y][xx * INPAINT_SMEM_TILE_X + threadIdx.x];
            dst[index] =
                center + 0.2f * (
                  smem[yy * INPAINT_SMEM_TILE_Y + threadIdx.y + 1][xx * INPAINT_SMEM_TILE_X + threadIdx.x]
                + smem[yy * INPAINT_SMEM_TILE_Y + threadIdx.y - 1][xx * INPAINT_SMEM_TILE_X + threadIdx.x]
                + smem[yy * INPAINT_SMEM_TILE_Y + threadIdx.y][xx * INPAINT_SMEM_TILE_X + threadIdx.x + 1]
                + smem[yy * INPAINT_SMEM_TILE_Y + threadIdx.y][xx * INPAINT_SMEM_TILE_X + threadIdx.x - 1]
                - 4.0f * center)
                - 0.2f * mask[index] * (center - src[index]);
        }
    }
}

__global__ void pdeHeatDiffusionKernel(bool* mask, float* src, float* tempSrc, int width, int height) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < 0 || y < 0 || x >= width || y >= height / INPAINT_UNROLL_Y) return;
    float center;
    int i;
    for (int k = 0; k < INPAINT_ITER_UNROLL; k++) {
#pragma unroll
        for (; y < height; y += height / INPAINT_UNROLL_Y) {
            center = tempSrc[y * width + x];
            i = y * width + x;
            tempSrc[i] = center
                + 0.2f
                * (tempSrc[max(0, y - 1) * width + x]
                    + tempSrc[min(height - 1, y + 1) * width + x]
                    + tempSrc[y * width + max(0, x - 1)]
                    + tempSrc[y * width + min(width - 1, x + 1)]
                    - 4.0f * center)
                - 0.2f * mask[i] * (center - src[i]);
        }
    }
}

#include "hairRemoval.cuh"


void HairRemoval::Process(cv::Mat& src, cv::Mat& dst) {
#if L2_TIMER
    auto t1 = getTime();
#endif
    _hairDetection(src);
#if L2_TIMER
    auto t2 = getTime();
#endif
    _hairInpainting(src, dst);
#if L2_TIMER
    auto t3 = getTime();
    printTime(t1, t2, "main -- detection");
    printTime(t2, t3, "main -- inpainting");
#endif
}

void HairRemoval::_hairDetection(cv::Mat& src) {
    HairDetectionInfo info = _detectionInfo;
    uchar* src_ptr = src.data;
    const int fftH = info.FFTH;
    const int fftW = info.FFTW;
    const unsigned long src_size = info.Width * info.Height * info.Channels;
    const unsigned long src_byte_size = src_size * sizeof(uchar);
    const unsigned long src_c_size = info.Width * info.Height;
    const unsigned long src_c_byte_size = src_c_size * sizeof(float);
    float* d_PaddedData;
    float* d_Kernel;
    float* d_PaddedKernel;
    float* d_DepthResult;
    float* d_src_c_ptr;
    uchar* d_Result;
    uchar* d_src_ptr;
    fComplex* d_DataSpectrum;
    fComplex* d_KernelSpectrum;
    fComplex* d_TempSpectrum;
    int* h_histogram = new int[DYNAMICRANGE];
    Check(hipMalloc((uchar**)&d_src_ptr, src_byte_size));
    Check(hipMalloc((uchar**)&(_detectionInfo.SplitSrc), src_byte_size));
    Check(hipMalloc((float**)&d_src_c_ptr, src_c_byte_size));
    Check(hipMalloc((void**)&d_Kernel, info.KernelH * info.KernelW * info.NumberOfFilter * sizeof(float)));
    Check(hipMalloc((void**)&d_PaddedData, fftH * fftW * sizeof(float)));
    Check(hipMalloc((void**)&d_PaddedKernel, fftH * fftW * sizeof(float)));
    Check(hipMalloc((void**)&d_DepthResult, fftH * fftW * info.NumberOfFilter * sizeof(float)));
    Check(hipMalloc((void**)&d_Result, info.Height * info.Width * sizeof(uchar)));
    Check(hipMalloc((void**)&(_detectionInfo.Mask), info.Height * info.Width * sizeof(bool)));
    Check(hipMalloc((void**)&d_DataSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    Check(hipMalloc((void**)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    Check(hipMalloc((void**)&d_TempSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));

    // init filter
    float* h_kernels = _initGaborFilterCube(info);

    // H to D
    Check(hipMemcpy(d_Kernel, h_kernels, info.KernelH * info.KernelW * info.NumberOfFilter * sizeof(float), hipMemcpyHostToDevice));
    Check(hipMemcpy(d_src_ptr, src_ptr, src_byte_size, hipMemcpyHostToDevice));
    dim3 block(DETECT_TILE_X, DETECT_TILE_Y / DETECT_UNROLL_Y);
    dim3 grid(iDivUp(info.Width, DETECT_TILE_X), iDivUp(info.Height, DETECT_TILE_Y));

    // only extract L channel
    extractLChannelKernel << < grid, block >> > (d_src_ptr, d_src_c_ptr, _detectionInfo.SplitSrc, info.Width, info.Height, info.Channels);
    Check(hipDeviceSynchronize());
    _padDataClampToBorder(d_PaddedData, d_src_c_ptr);

    // FFT data
    Check(hipfftExecR2C(_fftPlanFwd, (hipfftReal*)d_PaddedData, (hipfftComplex*)d_DataSpectrum));
    Check(hipDeviceSynchronize());
    for (int i = 0; i < info.NumberOfFilter; i++) {
        int kernel_offset = i * info.KernelH * info.KernelW;
        int data_offset = i * fftH * fftW;
        _padKernel(d_PaddedKernel, &(d_Kernel[kernel_offset]));

        // FFT kernel
        Check(hipfftExecR2C(_fftPlanFwd, (hipfftReal*)d_PaddedKernel, (hipfftComplex*)d_KernelSpectrum));
        Check(hipDeviceSynchronize());

        // mul
        _modulateAndNormalize(d_TempSpectrum, d_DataSpectrum, d_KernelSpectrum, 1);
        Check(hipfftExecC2R(_fftPlanInv, (hipfftComplex*)d_TempSpectrum, (hipfftReal*)(&d_DepthResult[data_offset])));
        Check(hipDeviceSynchronize());
    }
    _cubeReduction(d_DepthResult, d_Result);

    // histogram
    _makeHistogram(d_Result, h_histogram);
    _binarization(d_Result, _findOtsuThreshold(h_histogram));
    _dilation(d_Result, _detectionInfo.Mask);

#if PEEK_MASK
    bool* peek_mask = new bool[info.Width * info.Height];
    Check(hipMemcpy(peek_mask, _detectionInfo.Mask, info.Width * info.Height * sizeof(bool), hipMemcpyDeviceToHost));
    displayImage(peek_mask, info.Width, info.Height, "peek mask");
    delete[] peek_mask;
#endif

    // free
    delete[] h_histogram;
    Check(hipFree(d_Result));
    Check(hipFree(d_src_ptr));
    Check(hipFree(d_DataSpectrum));
    Check(hipFree(d_KernelSpectrum));
    Check(hipFree(d_PaddedData));
    Check(hipFree(d_PaddedKernel));
    Check(hipFree(d_TempSpectrum));
    Check(hipFree(d_src_c_ptr));
    Check(hipFree(d_Kernel));
    Check(hipFree(d_DepthResult));
}

void HairRemoval::_hairInpainting(cv::Mat& src, cv::Mat& dst) {
    HairInpaintInfo info = _inpaintInfo;
    float* d_masked_src;
    Check(hipMalloc((void**)&d_masked_src, info.NumberOfC3Elements * sizeof(float)));
    _normalizeImage(d_masked_src);
    _pdeHeatDiffusion(d_masked_src, dst.data);
    Check(hipFree(d_masked_src));
}

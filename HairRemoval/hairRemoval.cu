#include "hip/hip_runtime.h"
#include "hairRemoval.cuh"

HairRemoval::HairRemoval(int width, int height, int channel, bool isGPU) {
    _detectionInfo = HairDetectionInfo(width, height, channel, isGPU);
    _inpaintInfo = HairInpaintInfo(width, height, channel, isGPU);
}

void HairRemoval::Process(cv::Mat& src, cv::Mat& dst) {
    cv::Mat mask(cv::Size(src.cols, src.rows), CV_8U, cv::Scalar(0));
#if L2_TIMER
    auto t1 = getTime();
#endif
	_hairDetection(src, mask);
#if L2_TIMER
    auto t2 = getTime();
#endif
    cv::threshold(mask, mask, 0, 255, cv::THRESH_BINARY | cv::THRESH_OTSU);
#if L2_TIMER
    auto t3 = getTime();
#endif
    //_cleanIsolatedComponent(mask, hair_detection_info);
    cv::Mat kernel = cv::getStructuringElement(cv::MORPH_RECT, cv::Size(3, 3), cv::Point(-1, -1));
    cv::morphologyEx(mask, mask, cv::MORPH_DILATE, kernel, cv::Point(-1, -1), 1);
#if L2_TIMER
    auto t4 = getTime();
#endif
	_hairInpainting(src, mask, dst);
#if L2_TIMER
    auto t5 = getTime();
    printTime(t1, t2, "main -- detection", _detectionInfo.ExceedTime);
    printTime(t2, t3, "main -- entropyThesholding");
    printTime(t3, t4, "main -- cleanIsolatedComponent & morphology");
    printTime(t4, t5, "main -- inpainting");
#endif

#if PEEK_MASK
    displayImage(mask, "mask", false);
#endif
    gpuErrorCheck(hipDeviceReset());
}

void HairRemoval::_hairDetection(cv::Mat& src, cv::Mat& dst) {
    if (_detectionInfo.IsGPU) _hairDetectionGPU(src, dst);
    else _hairDetectionCPU(src, dst);
}

void HairRemoval::_hairDetectionGPU(cv::Mat& src, cv::Mat& dst) {
    HairDetectionInfo info = _detectionInfo;
    float* d_PaddedData;
    float* d_Kernel;
    float* d_PaddedKernel;
    float* d_DepthResult;
    float* d_src_c_ptr;
    uchar* d_Result;
    uchar* d_src_ptr;
    fComplex* d_DataSpectrum;
    fComplex* d_KernelSpectrum;
    fComplex* d_TempSpectrum;
    hipfftHandle fftPlanFwd;
    hipfftHandle fftPlanInv;
    
    uchar* src_ptr = src.data;
    const int depth = info.NumberOfFilter;
    const int fftH = snapTransformSize(info.Height + info.KernelH - 1);
    const int fftW = snapTransformSize(info.Width + info.KernelW - 1);
    const unsigned long src_size = src.cols * src.rows * src.channels();
    const unsigned long src_byte_size = src_size * sizeof(uchar);
    const unsigned long src_c_size = src.cols * src.rows;
    const unsigned long src_c_byte_size = src_c_size * sizeof(float);
    int block_x_size = TILE_DIM;
    int block_y_size = BLOCK_DIM;
    int grid_x_size = (src.cols + TILE_DIM - 1) / TILE_DIM;
    int grid_y_size = (src.rows + TILE_DIM - 1) / TILE_DIM;

    dim3 block(block_x_size, block_y_size);
    dim3 grid(grid_x_size, grid_y_size);

    // make a FFT plan (slow in CUDA 11)
    auto t1 = getTime();
    gpuErrorCheck(hipfftPlan2d(&fftPlanFwd, fftH, fftW, HIPFFT_R2C));
    gpuErrorCheck(hipfftPlan2d(&fftPlanInv, fftH, fftW, HIPFFT_C2R));
    auto t2 = getTime();
    _detectionInfo.ExceedTime = getDurationS(t1, t2);

    // allocate
    gpuErrorCheck(hipMalloc((uchar**)&d_src_ptr, src_byte_size));
    gpuErrorCheck(hipMalloc((float**)&d_src_c_ptr, src_c_byte_size));
    gpuErrorCheck(hipMalloc((void**)&d_Kernel, info.KernelH * info.KernelW * info.NumberOfFilter * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_PaddedData, fftH * fftW * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_PaddedKernel, fftH * fftW * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_DepthResult, fftH * fftW * info.NumberOfFilter * sizeof(float)));
    gpuErrorCheck(hipMalloc((void**)&d_Result, info.Height * info.Width * sizeof(uchar)));
    gpuErrorCheck(hipMalloc((void**)&d_DataSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    gpuErrorCheck(hipMalloc((void**)&d_KernelSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));
    gpuErrorCheck(hipMalloc((void**)&d_TempSpectrum, fftH * (fftW / 2 + 1) * sizeof(fComplex)));

    // init data
    float* h_kernels = _initGaborFilterCube(info);

    // H to D
    gpuErrorCheck(hipMemcpy(d_Kernel, h_kernels, info.KernelH * info.KernelW * info.NumberOfFilter * sizeof(float), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_src_ptr, src_ptr, src_byte_size, hipMemcpyHostToDevice));
    extractLChannelKernel << < grid, block>> > (d_src_ptr, d_src_c_ptr, src.cols, src.rows, src.channels());
    gpuErrorCheck(hipDeviceSynchronize());

    _padDataClampToBorder(d_PaddedData, d_src_c_ptr, fftH, fftW, info.Height, info.Width, info.KernelH, info.KernelW, info.KernelY, info.KernelX);

    // FFT data
    gpuErrorCheck(hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_PaddedData, (hipfftComplex*)d_DataSpectrum));
    gpuErrorCheck(hipDeviceSynchronize());

    for (int i = 0; i < info.NumberOfFilter; i++) {
        int kernel_offset = i * info.KernelH * info.KernelW;
        int data_offset = i * fftH * fftW;

        _padKernel(d_PaddedKernel, &(d_Kernel[kernel_offset]), fftH, fftW, info.KernelH, info.KernelW, info.KernelY, info.KernelX);

        // FFT kernel
        gpuErrorCheck(hipfftExecR2C(fftPlanFwd, (hipfftReal*)d_PaddedKernel, (hipfftComplex*)d_KernelSpectrum));
        gpuErrorCheck(hipDeviceSynchronize());

        // mul
        _modulateAndNormalize(d_TempSpectrum, d_DataSpectrum, d_KernelSpectrum, fftH, fftW, 1);
        gpuErrorCheck(hipfftExecC2R(fftPlanInv, (hipfftComplex*)d_TempSpectrum, (hipfftReal*)(&d_DepthResult[data_offset])));
        gpuErrorCheck(hipDeviceSynchronize());
    }
    _cubeReduction(d_DepthResult, d_Result, fftH, fftW, info.Height, info.Width, depth);

    // D to H and sync
    gpuErrorCheck(hipMemcpy(dst.data, d_Result, info.Height * info.Width * sizeof(uchar), hipMemcpyDeviceToHost));

    // free
    gpuErrorCheck(hipfftDestroy(fftPlanInv));
    gpuErrorCheck(hipfftDestroy(fftPlanFwd));
    gpuErrorCheck(hipFree(d_src_ptr));
    gpuErrorCheck(hipFree(d_DataSpectrum));
    gpuErrorCheck(hipFree(d_KernelSpectrum));
    gpuErrorCheck(hipFree(d_PaddedData));
    gpuErrorCheck(hipFree(d_PaddedKernel));
    gpuErrorCheck(hipFree(d_TempSpectrum));
    gpuErrorCheck(hipFree(d_src_c_ptr));
    gpuErrorCheck(hipFree(d_Kernel));
    gpuErrorCheck(hipFree(d_DepthResult));
}

void HairRemoval::_hairDetectionCPU(cv::Mat& src, cv::Mat& dst) {
    cv::Mat chL(cv::Size(src.cols, src.rows), CV_8U);
    _extractLChannel(src, chL);
    _gaborFiltering(chL, dst);
}

void HairRemoval::_gaborFiltering(cv::Mat& src, cv::Mat& dst) {
    const int rows = src.rows;
    const int cols = src.cols;
    const int depth = _detectionInfo.NumberOfFilter;
    const int step = src.channels();
    uchar* cube = new uchar[rows * cols * depth];

    // filtering
#pragma omp parallel for 
    for (int curNum = 0; curNum < depth; curNum++) {
        double theta = CV_PI / depth * curNum;
        cv::Mat kernel, tmp;
        kernel = _getGaborFilter(theta);

        filter2D(src, tmp, CV_8U, kernel); // tmp.type() == CV_8U

        // put AfterFilter into a cube
        int count = 0;
        int idx_for_cube = 0;

        uchar* tmpPtr = tmp.data;
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                idx_for_cube = curNum + depth * count;
                cube[idx_for_cube] = *tmpPtr;

                tmpPtr++;
                count++;
            }
        }
    }

    // max value
#pragma omp parallel for 
    for (int count = 0; count < rows * cols; count++) {
        int rRow = count / cols;
        int rCol = count % cols;
        int output_offset = rRow * cols + rCol;
        uchar* outPtr = dst.data + output_offset;

        uchar* start_element = cube + output_offset * depth;
        for (uchar* p = start_element; p != start_element + depth; p++) {
            if (*p > * outPtr) {
                *outPtr = *p;
            }
        }
    }
}

cv::Mat HairRemoval::_getGaborFilter(float theta) {
    HairDetectionInfo info = _detectionInfo;
    cv::Mat output(cv::Size(info.KernelRadius * 2 + 1, info.KernelRadius * 2 + 1), CV_64F, cv::Scalar(0.0));
    double* outPtr = (double*)output.data;
    for (int y = -info.KernelRadius; y < info.KernelRadius + 1; y++) {
        for (int x = -info.KernelRadius; x < info.KernelRadius + 1; x++, outPtr++) {
            double xx = x;
            double yy = y;
            double xp = xx * cos(theta) + yy * sin(theta);
            double yp = yy * cos(theta) - xx * sin(theta);
            *outPtr = exp(-CV_PI * (xp * xp / info.SigmaX / info.SigmaX + yp * yp / info.SigmaY / info.SigmaY)) * cos(CV_2PI * info.Beta / info.HairWidth * xp + CV_PI);
        }
    }
    return output;
}

void HairRemoval::_hairInpainting(cv::Mat& src, cv::Mat& mask, cv::Mat& dst) {
    HairInpaintInfo info = _inpaintInfo;
    cv::resize(src, src, cv::Size(info.Width, info.Height));
    cv::resize(mask, mask, cv::Size(info.Width, info.Height));
    float* raw_dst = (float*)malloc(info.NumberOfC3Elements * sizeof(float));
    float* normalized_src = (float*)malloc(info.NumberOfC3Elements * sizeof(float));
    float* normalized_mask = (float*)malloc(info.NumberOfC1Elements * sizeof(float));
    float* normalized_masked_src = (float*)malloc(info.NumberOfC3Elements * sizeof(float));
    _normalizeImage(src, mask, normalized_src, normalized_mask, normalized_masked_src);
    uchar* h_dst_RGB_array = (uchar*)malloc(info.NumberOfC3Elements * sizeof(uchar));
    if (info.IsGPU)
        _hairInpaintingGPU(normalized_mask, normalized_masked_src, raw_dst);
    else 
        _hairInpaintingCPU(normalized_mask, normalized_masked_src, raw_dst);
    _convertToMatArrayFormat(raw_dst, h_dst_RGB_array);
    cv::Mat dst_mat(info.Height, info.Width, CV_8UC3, h_dst_RGB_array);
    cv::resize(dst_mat, dst_mat, cv::Size(info.Width * info.RescaleFactor, info.Height * info.RescaleFactor));
    dst = dst_mat;

    free(normalized_src);
    free(normalized_mask);
    free(normalized_masked_src);
}

float* HairRemoval::_initGaborFilterCube(HairDetectionInfo para) {
    float* output = new float[para.KernelW * para.KernelH * para.NumberOfFilter];
    float* output_ptr = output;
    for (int curNum = 0; curNum < para.NumberOfFilter; curNum++) {
        float theta = (float)CV_PI / para.NumberOfFilter * curNum;
        for (int y = -para.KernelRadius; y < para.KernelRadius + 1; y++) {
            for (int x = -para.KernelRadius; x < para.KernelRadius + 1; x++, output_ptr++) {
                float xx = x;
                float yy = y;
                float xp = xx * cos(theta) + yy * sin(theta);
                float yp = yy * cos(theta) - xx * sin(theta);
                *output_ptr = exp((float)(-CV_PI) * (xp * xp / para.SigmaX / para.SigmaX + yp * yp / para.SigmaY / para.SigmaY)) * cos((float)CV_2PI * para.Beta / para.HairWidth * xp + (float)CV_PI);
            }
        }
    }
    return output;
}

void HairRemoval::_extractLChannel(cv::Mat& src, cv::Mat& dst) {
    uchar* src_ptr = src.data;
    uchar* dst_ptr = dst.data;
    int n_channels = src.channels();

#pragma omp parallel for
    for (int x = 0; x < src.cols; x++) {
        for (int y = 0; y < src.rows; y++) {
            uchar R = *(src_ptr + (y * src.step) + (x * n_channels) + 0);
            uchar G = *(src_ptr + (y * src.step) + (x * n_channels) + 1);
            uchar B = *(src_ptr + (y * src.step) + (x * n_channels) + 2);
            float l;
            float a;
            float b;
            RGBtoLab(R, G, B, l, a, b);
            *(dst_ptr + y * src.cols + x) = (uchar)l;
        }
    }
}

void HairRemoval::_padDataClampToBorder(float* d_Dst, float* d_Src, int fftH, int fftW, int dataH, int dataW, int kernelW, int kernelH, int kernelY, int kernelX) {
    assert(d_Src != d_Dst);
    dim3 block(TILE_DIM, TILE_DIM);
    dim3 grid(iDivUp(fftW, block.x), iDivUp(fftH, block.y));
    padDataClampToBorderKernel << <grid, block >> > (
        d_Dst,
        d_Src,
        fftH,
        fftW,
        dataH,
        dataW,
        kernelH,
        kernelW,
        kernelY,
        kernelX
        );
    getLastCudaError("padDataClampToBorder_kernel<<<>>> execution failed\n");
}

void HairRemoval::_padKernel(float* d_Dst, float* d_Src, int fftH, int fftW, int kernelH, int kernelW, int kernelY, int kernelX) {
    assert(d_Src != d_Dst);
    dim3 block(TILE_DIM, TILE_DIM);
    dim3 grid(iDivUp(kernelW, block.x), iDivUp(kernelH, block.y));
    padKernelKernel << <grid, block >> > (
        d_Dst,
        d_Src,
        fftH,
        fftW,
        kernelH,
        kernelW,
        kernelY,
        kernelX
        );
    getLastCudaError("padKernel_kernel<<<>>> execution failed\n");
    hipDeviceSynchronize();
}

void HairRemoval::_modulateAndNormalize(fComplex* d_Dst, fComplex* d_DataSrc, fComplex* d_KernelSrc, int fftH, int fftW, int padding) {
    assert(fftW % 2 == 0);
    const int dataSize = fftH * (fftW / 2 + padding);
    modulateAndNormalizeKernel << <iDivUp(dataSize, 256), 256 >> > (
        d_Dst,
        d_DataSrc,
        d_KernelSrc,
        dataSize,
        1.0f / (float)(fftW * fftH)
        );
    getLastCudaError("modulateAndNormalize() execution failed\n");
}

void HairRemoval::_cubeReduction(float* d_Src, uchar* d_Dst, int fftH, int fftW, int dataH, int dataW, int depth) {
    dim3 block(TILE_DIM, 8);
    dim3 grid(iDivUp(dataW, block.x), iDivUp(dataH, block.y));
    cubeReductionKernel << <grid, block >> > (
        d_Src,
        d_Dst,
        fftH,
        fftW,
        dataH,
        dataW,
        depth
        );
    getLastCudaError("CubeReductionKernel<<<>>> execution failed\n");
}

void HairRemoval::_cleanIsolatedComponent(cv::Mat& src) {
    cv::Mat labels, labels_uint8, stats, centroids;
    HairDetectionInfo info = _detectionInfo;
    std::vector<int> label_to_stay = std::vector<int>();

    int components = cv::connectedComponentsWithStats(src, labels, stats, centroids);
    int* statsPtr = (int*)stats.data;

    for (int i = 1; i < components; i++) {
        statsPtr = (int*)stats.data + i * stats.cols;
        int big_boundary = std::max(*(statsPtr + cv::CC_STAT_WIDTH), *(statsPtr + cv::CC_STAT_HEIGHT));
        int small_boundary = std::min(*(statsPtr + cv::CC_STAT_WIDTH), *(statsPtr + cv::CC_STAT_HEIGHT));
        int area = *(statsPtr + cv::CC_STAT_AREA);
        double ratio = (double)big_boundary / (double)small_boundary;

        if ((area > info.MinArea)) {
            label_to_stay.push_back(i);
        }
    }

    cv::Mat dst(cv::Size(src.cols, src.rows), CV_8U, cv::Scalar(0));
    cv::Mat look_up_table(cv::Size(1, DYNAMICRANGE), CV_8U, cv::Scalar(0));
    uchar* lutPtr = look_up_table.data;

    for (int i = 0; i < label_to_stay.size(); i++) {
        *(lutPtr + label_to_stay[i]) = DYNAMICRANGE - 1;
    }

    labels.convertTo(labels_uint8, CV_8U);
    cv::LUT(labels_uint8, look_up_table, dst);

    src = dst;
}

void HairRemoval::_normalizeImage(cv::Mat& srcImage, cv::Mat& srcMask, float* dstImage, float* dstMask, float* dstMaskImage) {
    HairInpaintInfo info = _inpaintInfo;
    const int width = srcImage.cols;
    const int height = srcImage.rows;
    uchar* src_image_ptr = srcImage.data;
    uchar* src_mask_ptr = srcMask.data;
#pragma omp parallel for
    for (int i = 0; i < height * width; i++) {
        dstMask[i] = src_mask_ptr[i] != 0 ? 0.0f : 1.0f;
    }
    int pixel = 0;
    int index = 0;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            index = y * (width * 3) + (x * 3);
            for (int k = 0; k < 3; k++) {
                pixel = src_image_ptr[index + k];
                if (pixel > info.MaxRgb[k]) info.MaxRgb[k] = pixel;
                if (pixel < info.MinRgb[k]) info.MinRgb[k] = pixel;
            }
        }
    }
    int range_list[] = { info.MaxRgb[0] - info.MinRgb[0], info.MaxRgb[1] - info.MinRgb[1], info.MaxRgb[2] - info.MinRgb[2] };
    for (int k = 0; k < 3; k++) {
        int channel_offset = k * width * height;
#pragma omp parallel for collapse (2)
        for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x++) {
                int maskI = y * width + x;
                int srcI = y * (width * 3) + (x * 3) + k;
                int dstI = channel_offset + maskI;
                float value = ((float)src_image_ptr[srcI] - info.MinRgb[k]) / range_list[k];
                dstImage[dstI] = value;
                dstMaskImage[dstI] = dstMask[maskI] > 0.0f ? value : 1.0f;
            }
        }

        for (int x = 0; x < width; x += width - 1) {
            for (int y = 0; y < height; y++) {
                int maskI = y * width + x;
                int dstI = channel_offset + maskI;
                dstMaskImage[dstI] = dstImage[dstI];
            }
        }
        for (int y = 0; y < height; y += height - 1) {
            for (int x = 0; x < width; x++) {
                int maskI = y * width + x;
                int dstI = channel_offset + maskI;
                dstMaskImage[dstI] = dstImage[dstI];
            }
        }
    }
}

void HairRemoval::_hairInpaintingGPU(float* normalized_mask, float* normalized_masked_src, float* dst) {
    HairInpaintInfo info = _inpaintInfo;
    float* d_normalized_mask;
    float* d_normalized_masked_src;
    float* d_normalized_masked_src_temp;
    gpuErrorCheck(hipMalloc((float**)&d_normalized_mask, info.NumberOfC1Elements * sizeof(float)));
    gpuErrorCheck(hipMalloc((float**)&d_normalized_masked_src, info.NumberOfC3Elements * sizeof(float)));
    gpuErrorCheck(hipMalloc((float**)&d_normalized_masked_src_temp, info.NumberOfC3Elements * sizeof(float)));
    gpuErrorCheck(hipMemcpy(d_normalized_mask, normalized_mask, info.NumberOfC1Elements * sizeof(float), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_normalized_masked_src, normalized_masked_src, info.NumberOfC3Elements * sizeof(float), hipMemcpyHostToDevice));
    gpuErrorCheck(hipMemcpy(d_normalized_masked_src_temp, d_normalized_masked_src, info.NumberOfC3Elements * sizeof(float), hipMemcpyDeviceToDevice));

    _pdeHeatDiffusionGPU(d_normalized_mask, d_normalized_masked_src, d_normalized_masked_src_temp);
    //_pdeHeatDiffusionSmemGPU(d_normalized_mask, d_normalized_masked_src, d_normalized_masked_src_temp);
    gpuErrorCheck(hipMemcpy(dst, d_normalized_masked_src_temp, info.NumberOfC3Elements * sizeof(float), hipMemcpyDeviceToHost));

    gpuErrorCheck(hipFree(d_normalized_mask));
    gpuErrorCheck(hipFree(d_normalized_masked_src));
    gpuErrorCheck(hipFree(d_normalized_masked_src_temp));
}

void HairRemoval::_pdeHeatDiffusionGPU(float* d_normalized_mask, float* d_normalized_masked_src, float* d_dst) {
    HairInpaintInfo info = _inpaintInfo;
    dim3 block(TILE_DIM, TILE_DIM, 1);
    dim3 grid(iDivUp(info.Width, TILE_DIM), iDivUp(info.Height, TILE_DIM), info.Channels);

    for (int i = 0; i < info.Iters; i++) {
        pdeHeatDiffusionKernel << <grid, block >> > (d_normalized_mask, d_normalized_masked_src, d_dst, info.Width, info.Height, info.Channels);
    }
}

void HairRemoval::_pdeHeatDiffusionSmemGPU(float* d_normalized_mask, float* d_normalized_masked_src, float* d_dst) {
    HairInpaintInfo info = _inpaintInfo;
    assert(info.Width / BlockDim_x * Step == 0);
    assert(info.Height / BlockDim_y * Step == 0);
    dim3 block(BlockDim_x, BlockDim_y);
    dim3 grid(iDivUp(info.Width, BlockDim_x * Step), iDivUp(info.Height, BlockDim_y * Step));

    for (int k = 0; k < info.Channels; k++) {
        float* src = d_normalized_masked_src + k * info.Width * info.Height;
        float* dst = d_dst + k * info.Width * info.Height;
        for (int i = 0; i < info.Iters; i++) {
            pdeHeatDiffusionSMEMKernel << <grid, block >> > (d_normalized_mask, src, dst, info.Width, info.Height);
        }
    }
}

void HairRemoval::_hairInpaintingCPU(float* normalized_mask, float* normalized_masked_src, float* dst) {
    HairInpaintInfo info = _inpaintInfo;
    memcpy(dst, normalized_masked_src, info.NumberOfC3Elements * sizeof(float));
    _pdeHeatDiffusionCPU(normalized_mask, normalized_masked_src, dst, info.Channels);
}

void HairRemoval::_pdeHeatDiffusionCPU(float* normalized_mask, float* normalized_masked_src, float* dst, int ch) {
    HairInpaintInfo info = _inpaintInfo;
    int x_boundary = info.Width - 1;
    for (int i = 0; i < info.Iters; i++) {
        for (int k = 0; k < ch; k++) {
            int channel_offset = k * info.Width * info.Height;
#pragma omp parallel for
            for (int y = 1; y < info.Height - 1; y++) {
#if ISAVX
                __m256 _pA = SET8F(0.0f);
                __m256 _pC = SET8F(0.0f);
                __m256 _mA = SET8F(0.0f);
                __m256 _mC = SET8F(0.0f);
                __m256 _eA = SET8F(0.0f);
                __m256 _eC = SET8F(0.0f);
                __m256 _dt = SET8F(info.Dt);
                __m256 _cw = SET8F(info.Cw);
                __m256 _x;
                __m256 _c, _u, _d, _l, _r, _mc, _oc;
                __m256i _x_mask;
                for (int x = 1; x < x_boundary; x += 8) {
                    int c1i = y * info.Width + x;
                    int c3i = channel_offset + c1i;
                    int c3ui = channel_offset + (y - 1) * info.Width + x;
                    int c3di = channel_offset + (y + 1) * info.Width + x;
                    int c3li = channel_offset + y * info.Width + (x - 1);
                    int c3ri = channel_offset + y * info.Width + (x + 1);

                    _x = SET8FE(x + 7.0f, x + 6.0f, x + 5.0f, x + 4.0f, x + 3.0f, x + 2.0f, x + 1.0f, x);
                    _x_mask = GETMASK(_x, SET8F(x_boundary));

                    _c = SET8F(0.0f);
                    _u = SET8F(0.0f);
                    _d = SET8F(0.0f);
                    _l = SET8F(0.0f);
                    _r = SET8F(0.0f);
                    _mc = SET8F(0.0f);
                    _oc = SET8F(0.0f);
                    _c = MASKLOAD(&dst[c3i], _x_mask);
                    _u = MASKLOAD(&dst[c3ui], _x_mask);
                    _d = MASKLOAD(&dst[c3di], _x_mask);
                    _l = MASKLOAD(&dst[c3li], _x_mask);
                    _r = MASKLOAD(&dst[c3ri], _x_mask);
                    _mc = MASKLOAD(&normalized_mask[c1i], _x_mask);
                    _oc = MASKLOAD(&normalized_masked_src[c3i], _x_mask);
                    MASKSTORE(&dst[c3i]
                        , _x_mask
                        , SUB8F(ADD8F(_c, MUL8F(_dt, SUB8F(ADD8F(_u, ADD8F(_d, ADD8F(_l, _r))), MUL8F(_cw, _c)))), MUL8F(_dt, MUL8F(_mc, SUB8F(_c, _oc)))));
                }
#else
                for (int x = 1; x < x_boundary; x++) {
                    int c1i = y * info.Width + x;
                    int c3i = channel_offset + c1i;
                    int c3ui = channel_offset + (y - 1) * info.Width + x;
                    int c3di = channel_offset + (y + 1) * info.Width + x;
                    int c3li = channel_offset + y * info.Width + (x - 1);
                    int c3ri = channel_offset + y * info.Width + (x + 1);

                    dst[c3i] = dst[c3i]
                        + info.Dt * (dst[c3ui] + dst[c3di] + dst[c3li] + dst[c3ri] - info.Cw * dst[c3i])
                        - info.Dt * normalized_mask[c1i] * (dst[c3i] - normalized_masked_src[c3i]);
                }
#endif
            }
        }
    }
}

void HairRemoval::_convertToMatArrayFormat(float* srcImage, uchar* dstImage) {
    HairInpaintInfo info = _inpaintInfo;
    for (int k = 0; k < info.Channels; k++) {
        int channel_offset = k * info.Width * info.Height;
        int range = info.MaxRgb[k] - info.MinRgb[k];
        int offset = info.MinRgb[k];
#pragma omp parallel for collapse (2)
        for (int y = 0; y < info.Height; y++) {
            for (int x = 0; x < info.Width; x++) {
                int dstI = y * (info.Width * info.Channels) + (x * info.Channels) + k;
                int srcI = channel_offset + y * info.Width + x;
                dstImage[dstI] = (uchar)(range * srcImage[srcI] + offset);
            }
        }
    }
}


